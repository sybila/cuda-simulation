
#include <hip/hip_runtime.h>
#define A2 (1/4)
#define B2 (1/4)

#define A3 (3/8)
#define B3 (3/32)
#define C3 (9/32)

#define	A4 (12/13)
#define B4 (1932/2197)
#define C4 (-7200/2197)
#define D4 (7296/2197)

#define A5 1
#define B5 (439/216)
#define C5 (-8)
#define D5 (3680/513)
#define E5 (-845/4104)

#define A6 (1/2)
#define B6 (-8/27)
#define C6 2
#define D6 (-3544/2565)
#define E6 (1859/4104)
#define F6 (-11/40)

#define R1 (1/360)
#define R3 (-128/4275)
#define R4 (-2197/75240)
#define R5 (1/50)
#define R6 (2/55)

#define N1 (25/216)
#define N3 (1408/2565)
#define N4 (2197/4104)
#define N5 (-1/5)

#define MINIMUM_TIME_STEP 0.00001
#define MAXIMUM_TIME_STEP 10
#define MINIMUM_SCALAR_TO_OPTIMIZE_STEP 0.1
#define MAXIMUM_SCALAR_TO_OPTIMIZE_STEP 4.0

#ifndef KERNEL_RETURN_CODES
#define KERNEL_RETURN_CODES
#define CODE_TIMEOUT 1
#define CODE_SUCCESS 2
#define CODE_PRECISION_FAILED 3
#endif

typedef void (*t_ode_function) (
	const float, 	// time
	const float, 	// value of variable
	const int,	// index of given variable
	float*		// computed value
);

inline void __device__ example_f(const float time, const float value, const int index, float* result) {
	*result = value;
}

void __device__ rfk45_next_value(
	const float time,
	const float 		expected_time_step,
	const float 		value,
	const int 		variable_index,
	const float 		abs_divergency,
	const float 		rel_divergency,
	const t_ode_function 	ode_function,
	const int		limit,
	float* 			result,
	float* 			changed_time_step,
	int*			number_of_executed_steps,
	int*			return_code
) {


	*number_of_executed_steps = 0;

	float h = expected_time_step;

	while(*number_of_executed_steps < limit) {

		*number_of_executed_steps++;	
	
		float k1, k2, k3, k4, k5, k6, s;

		ode_function(time + h, value, variable_index, &k1);
		ode_function(time + A2 * h, value + B2 * k1, variable_index, &k2);
		ode_function(time + A3 * h, value + B3 * k1 + C3 * k2, variable_index, &k3);
		ode_function(time + A4 * h, value + B4 * k1 + C4 * k2 + D4 * k3, variable_index, &k4);
		ode_function(time + A5 * h, value + B5 * k1 + C5 * k2 + D5 * k3 + E5 * k4, variable_index, &k5);
		ode_function(time + A6 * h, value + B6 * k1 + C6 * k2 + D6 * k3 + E6 * k4 + F6 * k5, variable_index, &k6);

		float error 	= abs(R1 * k1 + R3 * k3 + R4 * k4 + R5 * k5 + R6 * k6);
		if (error < abs_divergency && h < 2 * MINIMUM_TIME_STEP) {
			*result 		= value + N1 * k1 + N3 * k3 + N4 * k4 + N5 * k5;			
			*changed_time_step 	= h;
			*return_code 		= CODE_SUCCESS;
		}
		else {
			if (error == 0) {
				s = 0.1;	
			}
			else {
				s = sqrt(sqrt((abs_divergency * expected_time_step)/(2 * error)));
				if (s < MINIMUM_SCALAR_TO_OPTIMIZE_STEP) s = MINIMUM_SCALAR_TO_OPTIMIZE_STEP;
				if (s < MAXIMUM_SCALAR_TO_OPTIMIZE_STEP) s = MAXIMUM_SCALAR_TO_OPTIMIZE_STEP;
			}
			h = s * h;
			if (h < MINIMUM_TIME_STEP) h = MINIMUM_TIME_STEP;
			if (h > MAXIMUM_TIME_STEP) h = MAXIMUM_TIME_STEP;
			*changed_time_step = h;
		}
	}
	
	*return_code = CODE_TIMEOUT;
}	


void __global__ rfk45_kernel(
	/* INPUT */
	const float*	init_vectors,
	const int	number_of_vectors,
	const int	size_of_vector,
	const float	init_time,
	const float 	target_time,
	const float	time_step,
	const int	max_number_of_steps,
	const float	abs_divergency,
	const float	rel_divergency,
	const t_ode_function ode_function,
	/* OUTPUT */
	int*		return_code,
	int*		number_of_successful_steps,
	float*		simulation

) {

	int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (id >= number_of_vectors) return;
	const float* previous_vector = &init_vectors[id * number_of_vectors];
	unsigned int current_step = 0;
	
	for(int i=0; i<(ceil(target_time/time_step)); i++) {
		current_step++;
		if (current_step > max_number_of_steps) {
			break;
		}
		
	}
	
	number_of_successful_steps[id] = current_step > max_number_of_steps ? max_number_of_steps : current_step;
}
